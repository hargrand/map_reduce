﻿/**
 * @file map_reduce.cu
 */

#include "../spt/cuda_collection.cuh"
#include "../spt/support.hpp"

#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <string>

/**
 *  @brief Main entry point of the program.
 *  @details Demonstrates the usage of the Collection class and its map/reduce-style operations.
 *  @return 0 on successful execution.
 */
int main(int argc, char **argv)
{
	std::vector<std::string> args(argv, argv + argc);

	test_case tc = parse_args(args);
	std::vector<result<double>> results;

	hiprandState *d_rand_state;

	std::size_t max_size = tc.test_cases.empty()
							   ? 0
							   : *std::max_element(tc.test_cases.begin(), tc.test_cases.end());

	hipMalloc(&d_rand_state, max_size * sizeof(hiprandState));

	for (auto size : tc.test_cases)
	{
		// Initialize cuRAND states
		auto init_rand = [=] __device__(unsigned int i)
		{
			hiprand_init(1234, i, 0, &d_rand_state[i]);
			return hiprand_uniform_double(&d_rand_state[i]);
		};
		results.push_back(
			run_test<Collection<double>, double>(
				size,
				init_rand));
	}
	hipFree(d_rand_state);
	write_results(tc, results);

	return 0;
}
